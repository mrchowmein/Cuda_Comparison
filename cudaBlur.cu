#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include<time.h>


#define index(i, j, w)  ((i)*(w)) + (j)

__global__ void blurKernel (unsigned char *, unsigned char *, int, int, int);
void cudaBlur(unsigned char * , int, int, int);


int main(int argc, char * argv[]){

  unsigned char * imageArray;
  int w = (int) atoi(argv[1]);
  int h = (int) atoi(argv[2]);
  int numOfImages = (int) atoi(argv[3]);
  int blurSize = (int) atoi(argv[4]);


  imageArray = (unsigned char *)calloc(w*h, sizeof(unsigned char));

  //assign random unsigned chars to imageArray
  int i;
  for (i = 0; i <= w*h; i++) {
    imageArray[i] = (unsigned char) (rand() % 255 + 1);
  }



  int j;

  for(j = 0; j < numOfImages; j++){
    //printf("%u \n", imageArray[index(0,0,w)]);
    cudaBlur(imageArray, w, h, blurSize);
    //printf("%u \n", imageArray[0]);
  }



return 0;
}

void cudaBlur(unsigned char * imageArray, int w, int h, int blurSize)
{

  unsigned int num_bytes = w*h*sizeof(unsigned char);

  unsigned char * temp;
  temp = (unsigned char *)calloc(w*h, sizeof(unsigned char));

  memcpy((void *)temp, (void *) imageArray, num_bytes);



  //allocate device Memory
  unsigned char *d_inputArray;
  unsigned char *d_outputArray;
  hipMalloc(&d_inputArray, num_bytes);
  hipMalloc(&d_outputArray, num_bytes);

  dim3 threads_per_block( 128, 1, 1 );
  dim3 blocks_in_grid( ceil( (w*h)/ threads_per_block.x ), 1, 1 );

  clock_t t;
  t = clock();
  
  hipMemcpy(d_outputArray, temp, num_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_inputArray, imageArray, num_bytes, hipMemcpyHostToDevice);


  blurKernel<<<blocks_in_grid, threads_per_block>>>(d_inputArray, d_outputArray, w, h, blurSize);


  hipMemcpy(imageArray, d_outputArray, num_bytes, hipMemcpyDeviceToHost);
  t = clock() - t;
  double time_taken = ((double)t)/CLOCKS_PER_SEC;

  printf("kernel took %f seconds to execute \n", time_taken);
  //printf("done");
  //free device Memory
  hipFree(d_outputArray);
  hipFree(d_inputArray);
  free(temp);

}

__global__ void blurKernel (unsigned char * d_inputArray, unsigned char * d_outputArray,
 int w, int h, int blurSize){

    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    if(Col<w && Row < h){
      int pixVal = 0;
      int pixels = 0;

      for(int blurRow = -blurSize; blurRow < blurSize+1; ++blurRow){
        for(int blurCol = -blurSize; blurCol < blurSize+1; ++blurCol){
          int curRow = Row + blurRow;
          int curCol = Col + blurCol;

          //verify we have a valid image pixel
          if(curRow > -1 && curRow < h && curCol > -1 && curCol < w){
            pixVal += d_inputArray[curRow*w+curCol];
            pixels++; // keep track of number of pixels in the avg
          }
        }
      }

      //write our new pixel value out
      d_outputArray[Row*w+Col] = (unsigned char)(pixVal/pixels);


    }

}
